#include <iostream>
#include "hip/hip_runtime.h"

int main() {
    hipError_t cudaStatus;
    hipDeviceProp_t cuInfo; // gpu属性的类
    int l32count;
    cudaStatus = hipGetDeviceCount( &l32count); // 获取GPU数量
    if(cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipGetDeviceCount failed!");
        return 1;
    }
    std::cout << "the number of gpu: " << l32count << std::endl;

    for(int i=0; i<l32count; ++i)
    {
        cudaStatus = hipGetDeviceProperties(&cuInfo, i); // 获取GPU信息
        if(cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipGetDeviceProperties failed!");
            return 1;
        }
        printf("Name: %s\n", cuInfo.name);
    }
    std::cout << "Hello, World!" << std::endl;
    return 0;
}

